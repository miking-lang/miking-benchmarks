#include "hip/hip_runtime.h"
/*
 *  models/Yap_Dengue.cu
 *
 *  Copyright (C) 2021 Viktor Senderov and Paper Authors
 *
 *  The Yap-Dengue epidemilogical model.  Input hardcoded for now.
 *
 */

#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

/* Test182_observations_t
   Type wrapper of the test observations that we are using for the paper. */
struct Test183_observations_t {
  static const int NUM_OBSERVATIONS = 183;
  const int cases[NUM_OBSERVATIONS] = {1, 2, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 2, 0, 0, 1, 0, 0, 2, 1, 4, 2, 3, 2, 2, 4, 1, 3, 3, 4, 3, 3, 9, 1, 1, 7, 5, 4, 1, 2, 4, 7, 3, 6, 6, 4, 8, 6, 7, 2, 6, 7, 5, 7, 9, 10, 14, 9, 4, 5, 7, 10, 11, 17, 6, 13, 13, 14, 13, 12, 12, 15, 16, 12, 14, 11, 17, 10, 10, 16, 12, 17, 29, 21, 21, 25, 17, 12, 18, 11, 12, 10, 18, 8, 14, 10, 15, 16, 8, 7, 5, 7, 5, 5, 6, 11, 10, 5, 4, 9, 6, 1, 6, 3, 6, 4, 3, 5, 1, 8, 2, 9, 4, 5, 4, 3, 3, 4, 4, 3, 3, 4, 5, 2, 5, 4, 2, 6, 4, 2, 0, 4, 2, 1, 1, 1, 2, 3, 3, 3, 0, 3, 2, 1, 0, 1, 0, 0, 1, 2, 2, 1, 0, 1, 1, 1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 0};
};

typedef Test183_observations_t y_obs_t;

BBLOCK_DATA(y_obs, y_obs_t, 1)

typedef struct {
  int dS[y_obs->NUM_OBSERVATIONS]; // Newly succeptible
  int dE[y_obs->NUM_OBSERVATIONS]; // Newly exposed (incubating)
  int dI[y_obs->NUM_OBSERVATIONS]; // Newly infectious
  int dR[y_obs->NUM_OBSERVATIONS]; // Newly recovered

  int s[y_obs->NUM_OBSERVATIONS]; //  succeptible
  int e[y_obs->NUM_OBSERVATIONS]; //  exposed (incubating)
  int i[y_obs->NUM_OBSERVATIONS]; //  infectious
  int r[y_obs->NUM_OBSERVATIONS]; //  recovered

  floating_t nu; // birth probability ν
  floating_t mu; // survival probability μ
  floating_t lambda; // exposure probability λ
  floating_t delta; // infection probability δ
  floating_t gamma; // recovery probability γ
} SEIRComponent;


/* SEIRTransfer

 * - t: Step number.
 * - τ: Number of trials that may result in exposure.
 *
 * `τ` is computed externally according to the interaction between two
 * populations, then `transfer()` called to update the state of the
 * population.
 */
BBLOCK_HELPER(SEIRTransfer, {
    /* total population */
    int n = pop->s[t - 1] + pop->e[t - 1] + pop->i[t - 1] + pop->r[t - 1];

    /* transfers */
    pop->dE[t] = SAMPLE(binomial, pop->lambda, tau);
    pop->dI[t] = SAMPLE(binomial, pop->delta, pop->e[t - 1]);
    pop->dR[t] = SAMPLE(binomial, pop->gamma, pop->i[t - 1]);

    pop->s[t] = pop->s[t - 1] - pop->dE[t];
    pop->e[t] = pop->e[t - 1] + pop->dE[t] - pop->dI[t];
    pop->i[t] = pop->i[t - 1] + pop->dI[t] - pop->dR[t];
    pop->r[t] = pop->r[t - 1] + pop->dR[t];

    /* survival; we assume that if the survival rate is set to one, what is
     * meant is "all survive" regardless of the population size, and so do
     * not evaluate these, ensuring we don't get -inf weights for mismatching
     * numbers of trials (population sizes) */
    if (pop->mu != 1.0) {
      pop->s[t] = SAMPLE(binomial, pop->mu, pop->s[t]);
      pop->e[t] = SAMPLE(binomial, pop->mu, pop->e[t]);
      pop->i[t] = SAMPLE(binomial, pop->mu, pop->i[t]);
      pop->r[t] = SAMPLE(binomial, pop->mu, pop->r[t]);
    }

    /* births */
    if (pop->nu != 0.0) {
      pop->dS[t] = SAMPLE(binomial, pop->nu, n);
      pop->s[t] = pop->s[t] + pop->dS[t];
    }

  }, void, SEIRComponent* pop, int t, int tau);

typedef short obsIdx_t;

struct progState_t {
  obsIdx_t t;

  SEIRComponent m; // Mosquito
  SEIRComponent h; // Human

  floating_t rho; // Probability of a human case being observed.
  int z = 0; // Latent aggregate number of cases since last observation.
};

INIT_MODEL(progState_t)


/*
 * simObservation
 */
BBLOCK(simObservation,
{
  int t = PSTATE.t;
  y_obs_t* y = DATA_POINTER(y_obs);

  if (PSTATE.t < y->NUM_OBSERVATIONS - 1) PSTATE.t = ++t;
  else {
    NEXT = NULL;
    return;
  }
  assert(t>=1);
  int n = PSTATE.h.s[t - 1] + PSTATE.h.e[t - 1] + PSTATE.h.i[t - 1] + PSTATE.h.r[t - 1];

  /* transition of human population */
  int tau_h = SAMPLE(binomial, 1.0 - exp(-PSTATE.m.i[t - 1]/ (floating_t) n), PSTATE.h.s[t - 1]);
  BBLOCK_CALL(SEIRTransfer, &PSTATE.h, t, tau_h);

  /* transition of mosquito population */
  int tau_m = SAMPLE(binomial, 1.0 - exp(-PSTATE.h.i[t - 1]/(floating_t) n), PSTATE.m.s[t - 1]);
  BBLOCK_CALL(SEIRTransfer, &PSTATE.m, t, tau_m);
  PSTATE.z = PSTATE.z + PSTATE.h.dI[t];

  if (y->cases[t] != -1) {
    OBSERVE(binomial, PSTATE.rho, PSTATE.z, y->cases[t]);
    PSTATE.z = 0;
  }
 })


/*
 * simYapDengue
 */
BBLOCK(simYapDengue,
{
  int n = 7370;
  int t = PSTATE.t;

  PSTATE.h.i[t] = SAMPLE(poisson, 5.0);
  PSTATE.h.i[t] = PSTATE.h.i[t] + 1;
  PSTATE.h.e[t] = SAMPLE(poisson, 5.0);

  PSTATE.h.r[t] =   floor(SAMPLE(uniform, 0, 1 + n - PSTATE.h.i[t] - PSTATE.h.e[t]));
  PSTATE.h.s[t] = n - PSTATE.h.e[t] - PSTATE.h.i[t] - PSTATE.h.r[t];

  PSTATE.h.dS[t] = 0;
  PSTATE.h.dE[t] = PSTATE.h.e[t];
  PSTATE.h.dI[t] = PSTATE.h.i[t];
  PSTATE.h.dR[t] = 0;

  floating_t u = SAMPLE(uniform, -1.0, 2.0);
  PSTATE.m.s[t] = floor(n*pow(10.0, u));
  PSTATE.m.e[t] = 0;
  PSTATE.m.i[t] = 0;
  PSTATE.m.r[t] = 0;

  PSTATE.m.dS[t] = 0;
  PSTATE.m.dE[t] = 0;
  PSTATE.m.dI[t] = 0;
  PSTATE.m.dR[t] = 0;

  /* observation */
  y_obs_t* y = DATA_POINTER(y_obs);
  PSTATE.z = PSTATE.z + PSTATE.h.dI[t];
  if (y->cases[t] != -1) {
    OBSERVE(binomial, PSTATE.rho, PSTATE.z, y->cases[t]);
    PSTATE.z = 0;
  }

  NEXT = simObservation;

 })


BBLOCK(initialization, {
    PSTATE.h.nu = 0.0;
    PSTATE.h.mu = 1.0;
    PSTATE.h.lambda = SAMPLE(beta, 1.0, 1.0);
    PSTATE.h.delta = SAMPLE(beta, 1.0 + 2.0/4.4, 3.0 - 2.0/4.4);
    PSTATE.h.gamma = SAMPLE(beta, 1.0 + 2.0/4.5, 3.0 - 2.0/4.5);

    PSTATE.m.nu = 1.0/7.0;
    PSTATE.m.mu = 6.0/7.0;
    PSTATE.m.lambda = SAMPLE(beta, 1.0, 1.0);
    PSTATE.m.delta = SAMPLE(beta,1.0 + 2.0/6.5, 3.0 - 2.0/6.5);
    PSTATE.m.gamma = 0.0;

    PSTATE.rho =  SAMPLE(beta, 1.0, 1.0);

    PSTATE.z = 0;
    PSTATE.t = 0;

    NEXT = simYapDengue;
    BBLOCK_CALL(NEXT, NULL);
})


MAIN({
    FIRST_BBLOCK(initialization);
    SMC(NULL)
})
