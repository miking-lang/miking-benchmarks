/**
 Example using CRBD 
 */

#include <stdio.h>
#include <string>
#include <fstream>

#include "inference/smc/smc.cuh"
#include "../tree-utils/tree_utils.cuh"
#include "utils/math.cuh"

typedef bisse32_tree_t tree_t;
// typedef primate_tree_t tree_t;
// typedef moth_div_tree_t tree_t;
// typedef Accipitridae_tree_t tree_t;

const floating_t k = 1.0;
const floating_t theta = 1.0;
const floating_t kMu = 1.0;
const floating_t thetaMu = 0.5;

const floating_t epsMin = 0.0;
const floating_t epsMax = 1.0;

const floating_t rhoConst = 1.0;

#include "crbd.cuh"

MAIN(
    ADD_BBLOCK(simCRBD)
    ADD_BBLOCK(simTree)
    ADD_BBLOCK(survivorshipBias)

    SMC(saveResults)
)
