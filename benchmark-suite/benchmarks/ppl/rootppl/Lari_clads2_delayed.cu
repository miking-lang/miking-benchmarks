#include "hip/hip_runtime.h"
/*
 * Delayed version of Clads2 model.
 * Both parameters are delayed.
 *
 */

#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "../tree-utils/tree_utils.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

typedef Lari_tree_t tree_t;
const floating_t rho = 0.8410596026490066;
  
const floating_t k = 1;
const floating_t theta = 1;
const floating_t kMu = 1;
const floating_t thetaMu = 0.5;

const floating_t m0 = 0;
const floating_t v = 1;
const floating_t a = 1.0;
const floating_t b = 0.2;

std::string analysisName = "exp-02";

#define NUM_BBLOCKS 5

#include "./clads2_delayed.cuh"

MAIN({

    ADD_BBLOCK(simClaDS2);
    ADD_BBLOCK(simTree);
    ADD_BBLOCK(conditionOnDetection);
    ADD_BBLOCK(justResample);
    ADD_BBLOCK(sampleFinalLambda);
    //SMC(saveResults);
    SMC(NULL)
})
