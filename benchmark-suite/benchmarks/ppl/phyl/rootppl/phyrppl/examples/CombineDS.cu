#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "trees/tree_utils.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"
#include "trees/cetaceans.cuh"
#include "trees/default_trees.cuh"

#define CLADS false              // Cladogenetic changes
#define ANADS true               // Anagenetic changes
#define EXTINCTION 2            // 2 - constant turnover, 1 - const, 0 - no exticntion

#define RARE_SHIFT false          // RARE_DS model TODO
#define RESAMPLE_RATES false      // TODO resample turnover and anagenesis rate at rate shifts

/* Do not tune unless you know what you're doing! */
#define GUARD true
#define NICOLAS false
#define MAX_FACTOR 1e5 
#define MIN_FACTOR 1e-5
#define M 20              // Number of subsamples to draw
#define DEBUG false
unsigned int depth;


//typedef bisse32_tree_t tree_t;
typedef cetaceans_87_tree_t tree_t;
BBLOCK_DATA(tree, tree_t, 1)
BBLOCK_DATA_CONST(rho, floating_t, 1.0)

#define NUM_BBLOCKS 4
#include "../models/CombineDS.cuh"

BBLOCK(initialization, {
    // Priors
    PSTATE.lambda_0 = gamma_t(1.0, 1.0);
    PSTATE.mu_0 = gamma_t(1.0, 0.5);
    PSTATE.nu_0 = gamma_t(1.0, 2.0);
    PSTATE.alpha_sigma = normalInverseGamma_t(0, 1.0, 1.0, 0.2);
    PSTATE.alpha_sigma_nu = normalInverseGamma_t(0, 1.0, 1.0, 0.2);
    
    //PSTATE.epsilon = 0.5;
    //PSTATE.ypsilon = 1;
    
    // // Immediate sampling
    // floating_t lambda0 = SAMPLE(gamma, lambda_0.k, lambda_0.theta);
    // floating_t sigma = sqrt( 1/ SAMPLE(gamma, a, 1/b));
    // floating_t alpha = exp( SAMPLE(normal, m0, sigma));
    
    PC++;
})

MAIN({
    ADD_BBLOCK(initialization);
    ADD_BBLOCK(simCombineDS);
    ADD_BBLOCK(simTree);
    //ADD_BBLOCK(conditionOnDetection);
    //ADD_BBLOCK(sampleFinalLambda);
    //SMC(saveResults);
    SMC(NULL)
})
