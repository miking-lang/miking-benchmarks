#include "hip/hip_runtime.h"
/* 
 *  models/CombineDS.cuh
 *
 *  Copyright (C) 2020-2021 Viktor Senderov, Joey Öhman, David Broman
 * 
 *
 *  CombineDS diversification model supports conditionally simulates
 *  several different types of evolution:
 *
 *    - cladogenetic (ClaDS-like) small changes in diversification
 *      rates, ClaDS versions 0-2.
 *	      
 *    - anagenetic small changes (happening on a single lineage)
 *
 *    - rare large shits (both anagenetic and cladogenetic)
 *
 *    - uncoupling of the turnover rate at the rare large shifts for
 *      ClaDS2
 *
 *
 *  This file needs to be included by a .cu file, containing the MAIN
 *  macro, needed global parameters, needed tuning parameters, and 
 *  the tree structure as a datatype.
 * 
 *  Needed global parameters:
 * 
 *    const floating_t k = 1;            // prior Γ-shape for λ
 *    const floating_t theta = 1;        // prior Γ-scale for λ
 *
 *    const floating_t kNu = 1;          // prior Γ-shape for ν
 *    const floating_t thetaNu = 0.5;    // prior Γ-shape for ν
 *
 *    const floating_t a_epsilon = 1;    // prior β-shape 1 for p_ε
 *    const floating_t b_epsilon = 100;  // prior β-shape 2 for p_ε
 *
 *    const floating_t m0 = 0;   // Hyper-param of prior for α and σ
 *    const floating_t v = 1;    // Hyper-param of prior for α and σ
 *    const floating_t a = 1.0;  // Hyper-param of prior for α and σ
 *    const floating_t b = 0.2;  // Hyper-param of prior for α and σ
 * 
 *  Needed tuning parameters:
 *
 *    #define M 20              // Number of subsamples to draw
 *    #define RARE_SHIFT false  // Activate rare shifts
 *    #define CLADS true        // Cladogenetic changes
 *    #define ANADS true        // Anagenetic changes
 *    #define UNCOUPLE true     // Uncouples turnover rate at rare shifts
 *    #define CLADS1 false      // ClaDS version: 0, 1, or 2, TODO 0
 *
 *  Tree selection, 3 steps:
 *
 *    #include "trees/cetaceans.cuh"       // (1)
 *    typedef cetaceans_87_tree_t tree_t;  // (2)
 *    const floating_t rhoConst = 1.00;    // (3) sampling rate
 *
 *  models/CombineDS.cuh defines the following BBLOCKS that can be included
 *  in the MAIN macro:
 *
 *    - simCombinedDS         (required)
 *
 *    - simTree               (required)
 *
 *    - conditionOnDetection  (optional, corrects for survivorship bias)
 *
 *    - sampleFinalLambda     (optional, samples the global parameters,
 *                             which have been delayed)
 *
 *    - saveResults           (optional callback, needs to be used in 
 *                             conjunction with sampleFinalLambda)
 */

/* Preamble */
#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "trees/tree_utils.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

#include "trees/cetaceans.cuh"
#include "trees/default_trees.cuh"



/* Tunable parameters */
#define CLADS false              // Cladogenetic changes
#define CONST_EXTINCTION false   // Constant extinction rate, if it is false CONSTANT_EXTINCTION
//#define ZERO_EXTINCTION        // TODO 

#define ANADS true               // Anagenetic changes
#define CONST_ANAGENESIS false  // TODO

#define RARE_SHIFT false          // Activate rare shifts - works both on ClaDS and AnaDS
#define UNCOUPLE false            // Uncouples turnover rate at rare shifts
// ?? Do we resample yspislon at rate shifts ??

/* Do not tune unless you know what you're doing! */
#define GUARD true
#define MAX_FACTOR 1e5 
#define MIN_FACTOR 1e-5
#define M 20              // Number of subsamples to draw
#define DEBUG false
#define DEBUG1 false
#define DEBUG2 false
unsigned int depth;

/* Tree selection */
//typedef cetaceans_87_tree_t tree_t;
typedef bisse32_tree_t tree_t;
const floating_t rhoConst = 1.0;

/* Priors for diversification parameters λ, μ, ν*/
const floating_t k = 1.0;
const floating_t theta = 1.0;
const floating_t epsilon = 0.5;   // initial extinction rate
const floating_t ypsilon = 0.5;   // initial anagenesis rate

/* Rare shift frequency */
const floating_t a_epsilon = 1;
const floating_t b_epsilon = 100;

/* Concept paper priors */
const floating_t m0 = 0;
const floating_t v = 1;
const floating_t a = 1.0;
const floating_t b = 0.002;

/* New, small shift priors */
// const floating_t m0 = 0;
// const floating_t v = 1;
// const floating_t a = 1.0;
//const floating_t b = 0.2;



/////////////////////////////////////////////////////////////////////////////



BBLOCK_DATA(tree, tree_t, 1)
BBLOCK_DATA_CONST(rho, floating_t, rhoConst)
typedef short treeIdx_t;

/* Program state */
struct progState_t {
  floating_t factors[(tree->NUM_NODES)] = {1.0}; // first is 1, all other 0 for now
  // TODO
  // Technically we don't need a factor for the root (it is assumed to be 1)
  // But for now we are going to waste one posistion for easier debugging.

  floating_t turnover_rates[(tree->NUM_NODES)]; // used to multiply the scale of μ

  bool cladsShifts[(tree->NUM_NODES)] = {0}; // initalize with 0 
  bool anadsShifts[(tree->NUM_NODES)] = {0};
  
  // Distributions, use underscores to denote distributions
  gamma_t lambda_0;
  gamma_t mu_0;
  gamma_t nu;
  normalInverseGamma_t alpha_sigma;
  beta_t ab;
  
  // Final Values/ Hyperparameters
  // TODO do we need all?
  floating_t lambda0;
  floating_t mu0;
  //  floating_t nu; // name clash
  floating_t alpha;
  floating_t sigma;
  floating_t epsilon;  // initial turn-over rate
  floating_t pEpsilon; // probability of large shift
  treeIdx_t treeIdx;
  int nshifts_clads;
  int nshifts_anads;
};


#define NUM_BBLOCKS 1

INIT_MODEL(progState_t, NUM_BBLOCKS)


/*
 * simCombineDS - required BBLOCK
 */
BBLOCK(simCombineDS,
{

  // Set up tree traversal
  tree_t* treeP = DATA_POINTER(tree);
  PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];
  
  // Draw initial rates, or delayed declare them

  gamma_t lambda_0(k, theta);
  gamma_t mu_0(k, theta);
  gamma_t nu(k, theta);
  beta_t ab(a_epsilon, b_epsilon);
  normalInverseGamma_t alpha_sigma(m0, v, a, b);

  PSTATE.epsilon = epsilon;  PSTATE.lambda_0 = lambda_0;
  PSTATE.mu_0 = mu_0;
  PSTATE.nu = nu;
  PSTATE.ab = ab;
  PSTATE.alpha_sigma = alpha_sigma;
  
  floating_t f1 = SAMPLE(sample_NormalInverseGammaNormal, PSTATE.alpha_sigma);


  floating_t a = SAMPLE(

  printf("%f %f\n", f1, exp(f1));
  // Advance to next BBLOCK
  PC++;
   
})



MAIN({
    ADD_BBLOCK(simCombineDS);
    SMC(NULL)
})
