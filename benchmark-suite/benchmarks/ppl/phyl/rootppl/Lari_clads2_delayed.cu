#include "hip/hip_runtime.h"
/*
 * Delayed version of Clads2 model.
 * Both parameters are delayed.
 *
 */

#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

#include "phyrppl/trees/tree_utils.cuh"
#include "phyrppl/trees/birds.cuh"

typedef Lari_tree_t tree_t;
const floating_t rho = 0.8410596026490066;
  
const floating_t k = 1;
const floating_t theta = 1;
const floating_t kMu = 1;
const floating_t thetaMu = 0.5;

const floating_t m0 = 0;
const floating_t v = 1;
const floating_t a = 1.0;
const floating_t b = 0.2;

std::string analysisName = "exp-03";
const int M = 25; // subsample size

#define NUM_BBLOCKS 3

#include "phyrppl/models/ClaDS2_delayed.cuh"

MAIN({
    FIRST_BBLOCK(simClaDS2)
    SMC(saveResults);
    //SMC(NULL)
})
